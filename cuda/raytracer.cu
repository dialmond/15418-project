#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <float.h>
#include <hiprand/hiprand_kernel.h>
#include "vec3.h"
#include "ray.h"
#include "sphere.h"
#include "hittable_list.h"
#include "camera.h"
#include "material.h"
#include "color.h"


#define DEBUG
#ifdef DEBUG
#define cudaCheckError(ans) cudaAssert((ans), __FILE__, __LINE__);
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "CUDA Error: %s at %s:%d\n",
                hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}
#else
#define cudaCheckError(ans) ans
#endif

__device__ float random_float_cuda(hiprandState* local_rand_state) {
    // Returns a random real in [0,1).
    return hiprand_uniform(local_rand_state);
}

__device__ vec3 sample_square(hiprandState* local_rand_state) {
    // Returns the vector to a random point in the [-.5,-.5]-[+.5,+.5] unit square.
    return vec3(random_float_cuda(local_rand_state) - 0.5, random_float_cuda(local_rand_state) - 0.5, 0);
}

__device__ point3 defocus_disk_sample(camera* cam, hiprandState* local_rand_state) {
    // Returns a random point in the camera defocus disk.
    auto p = random_in_unit_disk(local_rand_state);
    return cam->center + (p[0] * cam->defocus_disk_u) + (p[1] * cam->defocus_disk_v);
}

__device__ ray get_ray(int i, int j, camera* cam, hiprandState* local_rand_state) {
    // Construct a camera ray originating from the defocus disk and directed at a randomly
    // sampled point around the pixel location i, j.

    auto offset = sample_square(local_rand_state);
    auto pixel_sample = cam->pixel00_loc
                        + ((i + offset.x()) * cam->pixel_delta_u)
                        + ((j + offset.y()) * cam->pixel_delta_v);
    
    // if (i == 0 && j == 0) {
    //     printf("Pixel: %d, %d\n", i, j);
    //     printf("Pixel sample: %d\n", pixel_sample);
    //     printf("Pixel delta u: %f\n", cam->pixel_delta_u);
    //     printf("Pixel delta v: %f\n", cam->pixel_delta_v);
    //     printf("Offset: %d %d\n", offset.x(), offset.y());
    //     printf("Pixel 00 loc: %f\n", cam->pixel00_loc);
    // }

    auto ray_origin = (cam->defocus_angle <= 0) ? cam->center : defocus_disk_sample(cam, local_rand_state);
    auto ray_direction = pixel_sample - ray_origin;

    return ray(ray_origin, ray_direction);
}

__device__ color ray_color(ray& r, int depth, hittable** world, hiprandState* local_rand_state) {
    ray cur_ray = r;
    vec3 cur_attenuation = vec3(1.0, 1.0, 1.0);

    for (int i = 0; i < depth; i++) {
        hit_record rec;
        if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
            ray scattered;
            vec3 attenuation;
            if (rec.mat->scatter(cur_ray, rec, attenuation, scattered, local_rand_state)) {
                cur_attenuation = cur_attenuation * attenuation;
                cur_ray = scattered;
            } else {
                return color(0, 0, 0);
            }
        }
        else {
            vec3 unit_direction = unit_vector(r.direction());
            auto a = 0.5*(unit_direction.y() + 1.0);
            return cur_attenuation * ((1.0-a)*color(1.0, 1.0, 1.0) + a*color(0.5, 0.7, 1.0));
        }
    }
    return color(0, 0, 0); // exceeded recursion

}

__global__ void print_thread_indices() {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    printf("Thread index: (%d, %d, %d)\n", i, j, k);
}

__global__ void render_kernel(color* framebuffer,  hittable** world, camera** c) {
    printf("Inside render kernel\n");
    camera* cam = *c;
    // hittable_list* world = *w;
    int image_width = cam->image_width;
    int image_height = cam->image_height;
    int samples_per_pixel = cam->samples_per_pixel;
    int max_depth = cam->max_depth;

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    int pixel_index = j * image_width + i;


    if (i >= image_width || j >= image_height) return;

    // printf("Thread index: (%d, %d)\n", i, j);
    // printf("Pixel index: %d\n", pixel_index);

    // Initialize CURAND state
    hiprandState local_rand_state;
    hiprand_init(1984, pixel_index, 0, &local_rand_state); //1984 is the seed

    
    color pixel_color(0, 0, 0);
    for (int sample = 0; sample < samples_per_pixel; sample++) {
        ray r = get_ray(i, j, cam, &local_rand_state);
        // printf("Ray: %f %f %f\n", r.direction().x(), r.direction().y(), r.direction().z());
        pixel_color += ray_color(r, max_depth, world, &local_rand_state);
    // //     pixel_color += ray_color(r, max_depth, world);
        // printf("After ray color function\n");
        // printf("Pixel color: %f %f %f\n", pixel_color.x(), pixel_color.y(), pixel_color.z());
    // }
    }

    framebuffer[pixel_index] = pixel_color;
}

__global__ void createShapeObjects(hittable **d_list, hittable **d_world) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        d_list[0] = new sphere(vec3(0.0,-100.5,-1.0), 100.0,
                        new lambertian(vec3(0.8, 0.8, 0.0)));
        d_list[1] = new sphere(vec3(0,0,-1.2), 0.5,
                                new lambertian(vec3(0.1, 0.2, 0.5)));
        d_list[2] = new sphere(vec3(-1.0,0.0,-1.0), 0.5,
                                 new dielectric(1.50));
        d_list[3] = new sphere(vec3(-1.0,0,-1.0), 0.4,
                                 new dielectric(1.00/1.50));
        d_list[4] = new sphere(vec3(1,0,-1), 0.5,
                        new metal(vec3(0.8, 0.6, 0.2), 1.0));
        *d_world = new hittable_list(d_list,5);
    }
}

__global__ void createCamera(camera** cam, const camera& c) {
    *cam = new camera();

    (*cam)->aspect_ratio = 16.0 / 9.0;
    (*cam)->image_width  = 400;
    (*cam)->samples_per_pixel = 10; //was 100
    (*cam)->max_depth = 50;

    (*cam)->vfov     = 20;
    (*cam)->lookfrom = point3(-2,2,1);
    (*cam)->lookat   = point3(0,0,-1);
    (*cam)->vup      = vec3(0,1,0);

    (*cam)->defocus_angle = 10.0;
    (*cam)->focus_dist    = 3.4;
    (*cam)->initialize();
}

// __global__ void checkCamera(camera** cam) {
//     printf("Camera aspect ratio: %f\n", (*cam)->aspect_ratio);
//     printf("Camera image width: %d\n", (*cam)->image_width);
//     printf("Camera samples per pixel: %d\n", (*cam)->samples_per_pixel);
//     printf("Camera max depth: %d\n", (*cam)->max_depth);
// }


void intializeKernel(const camera& cam, int image_width, int image_height) {
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, 0);
    printf("Max Threads Per Block: %d\n", props.maxThreadsPerBlock);


    hittable** d_world;
    hittable** d_list;
    color* d_framebuffer;
    camera** d_camera;
    size_t framebuffer_size = image_width * image_height * sizeof(color);

    // cudaCheckError();
    // hipDeviceReset();

    std::clog << "Initializing CUDA kernel...\n";
    // Allocate memory for the framebuffer on the device
    cudaCheckError(hipMalloc(&d_framebuffer, framebuffer_size));

    // Allocate memory for the camera on the device
    cudaCheckError(hipMalloc((void **)&d_camera, sizeof(camera*)));
    createCamera<<<1,1>>>(d_camera, cam);
    cudaCheckError(hipDeviceSynchronize());

    // // Allocate memory for the world on the device
    // cudaCheckError(hipMalloc((void**)&d_list, 5 * sizeof(hittable*)));
    // cudaCheckError(hipMalloc((void**)&d_world, sizeof(hittable*)));

    // createShapeObjects<<<1,1>>>(d_list, d_world);
    // cudaCheckError(hipDeviceSynchronize());
    
    // // Define block and grid sizes
    // dim3 block_size(16, 16);
    // dim3 grid_size((image_width + block_size.x - 1) / block_size.x, 
    //         (image_height + block_size.y - 1) / block_size.y);

    // // Launch the CUDA kernel
    // std::clog << "Launching CUDA kernel...\n";
    // render_kernel<<<grid_size, block_size>>>(d_framebuffer, d_world, d_camera);
    // cudaCheckError(hipDeviceSynchronize());
    // std::clog << "CUDA kernel finished.\n";

    // // Allocate memory for the framebuffer on the host using malloc
    // color* h_framebuffer = (color*)malloc(image_width * image_height * sizeof(color));
    // hipMemcpy(h_framebuffer, d_framebuffer, framebuffer_size, hipMemcpyDeviceToHost);

    // // Free GPU memory
    // hipFree(d_framebuffer);
    // hipFree(d_world);
    // hipFree(d_camera);

    // Output the image
    // std::clog << "Rendering image...\n";
    // std::cout << "P3\n" << image_width << ' ' << image_height << "\n255\n";
    // for (int j = 0; j < image_height; j++) {
    //     for (int i = 0; i < image_width; i++) {
    //         write_color(std::cout, cam.pixel_samples_scale * h_framebuffer[j * image_width + i]);
    //     }
    // }
    // std::clog << "\nDone.\n";

    // Free the allocated memory
    // free(h_framebuffer);
}

void rayCUDA(){
    camera cam;

    cam.aspect_ratio = 16.0 / 9.0;
    cam.image_width  = 400;
    cam.samples_per_pixel = 10; //was 100
    cam.max_depth = 50;

    cam.vfov     = 20;
    cam.lookfrom = point3(-2,2,1);
    cam.lookat   = point3(0,0,-1);
    cam.vup      = vec3(0,1,0);

    cam.defocus_angle = 10.0;
    cam.focus_dist    = 3.4;

    cam.initialize();

    intializeKernel(cam, cam.image_width, cam.image_height);
}